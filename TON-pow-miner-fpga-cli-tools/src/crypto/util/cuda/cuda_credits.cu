#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "miner.h"
#include "cuda_helper.h"

#define ROTRIGHT(a, b) (((a) >> (b)) | ((a) << (32 - (b))))

#define CH(x, y, z) (((x) & (y)) ^ (~(x) & (z)))
#define MAJ(x, y, z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))
#define EP0(x) (ROTRIGHT(x, 2) ^ ROTRIGHT(x, 13) ^ ROTRIGHT(x, 22))
#define EP1(x) (ROTRIGHT(x, 6) ^ ROTRIGHT(x, 11) ^ ROTRIGHT(x, 25))
#define SIG0(x) (ROTRIGHT(x, 7) ^ ROTRIGHT(x, 18) ^ ((x) >> 3))
#define SIG1(x) (ROTRIGHT(x, 17) ^ ROTRIGHT(x, 19) ^ ((x) >> 10))

struct vec8u {
  uint32_t v[8];
};

struct vec16u {
  uint32_t v[16];
};

static const vec8u h256 = {0x6A09E667, 0xBB67AE85, 0x3C6EF372, 0xA54FF53A,
                           0x510E527F, 0x9B05688C, 0x1F83D9AB, 0x5BE0CD19};

static __forceinline__ __device__ __host__ vec8u sha256_transform(vec16u data, vec8u state) {
  const uint32_t ksha[] = {
      0x428A2F98, 0x71374491, 0xB5C0FBCF, 0xE9B5DBA5, 0x3956C25B, 0x59F111F1, 0x923F82A4, 0xAB1C5ED5,
      0xD807AA98, 0x12835B01, 0x243185BE, 0x550C7DC3, 0x72BE5D74, 0x80DEB1FE, 0x9BDC06A7, 0xC19BF174,
      0xE49B69C1, 0xEFBE4786, 0x0FC19DC6, 0x240CA1CC, 0x2DE92C6F, 0x4A7484AA, 0x5CB0A9DC, 0x76F988DA,
      0x983E5152, 0xA831C66D, 0xB00327C8, 0xBF597FC7, 0xC6E00BF3, 0xD5A79147, 0x06CA6351, 0x14292967,
      0x27B70A85, 0x2E1B2138, 0x4D2C6DFC, 0x53380D13, 0x650A7354, 0x766A0ABB, 0x81C2C92E, 0x92722C85,
      0xA2BFE8A1, 0xA81A664B, 0xC24B8B70, 0xC76C51A3, 0xD192E819, 0xD6990624, 0xF40E3585, 0x106AA070,
      0x19A4C116, 0x1E376C08, 0x2748774C, 0x34B0BCB5, 0x391C0CB3, 0x4ED8AA4A, 0x5B9CCA4F, 0x682E6FF3,
      0x748F82EE, 0x78A5636F, 0x84C87814, 0x8CC70208, 0x90BEFFFA, 0xA4506CEB, 0xBEF9A3F7, 0xC67178F2};

  uint32_t m[64];

  for (int i = 0; i < 16; i += 1)
    m[i] = data.v[i];

  for (int i = 16; i < 64; i += 1)
    m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];

  uint32_t a = state.v[0], b = state.v[1], c = state.v[2], d = state.v[3];
  uint32_t e = state.v[4], f = state.v[5], g = state.v[6], h = state.v[7];

#if defined(__CUDA_ARCH__)
#pragma unroll
#endif
  for (int i = 0; i < 64; i += 1) {
    uint32_t t1 = h + EP1(e) + CH(e, f, g) + ksha[i] + m[i];
    uint32_t t2 = EP0(a) + MAJ(a, b, c);
    h = g;
    g = f;
    f = e;
    e = d + t1;
    d = c;
    c = b;
    b = a;
    a = t1 + t2;
  }

  state.v[0] += a, state.v[1] += b, state.v[2] += c, state.v[3] += d;
  state.v[4] += e, state.v[5] += f, state.v[6] += g, state.v[7] += h;

  return state;
}

struct ThreadData {
  vec8u state;
  uint32_t rdata[9];
};

struct MsgData {
  uint32_t pseed[3];
  ThreadData thrdata[MAX_GPU_THREADS];
  vec8u target;
};

struct DevHashResult {
  uint64_t nonce;
  uint32_t vcpu, found;
};

__constant__ MsgData c_msg;

extern "C" __global__ void bitcredit_gpu_hash(uint64_t start_nonce, DevHashResult *result) {
  uint32_t vcpu = blockIdx.y;
  ThreadData *thrdata = &c_msg.thrdata[vcpu];

  vec8u state = thrdata->state;
  vec16u data;

  uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t nonce = start_nonce + idx;

  uint32_t rdata6 = thrdata->rdata[0], rdata7 = thrdata->rdata[1], rdata8 = thrdata->rdata[2];
  uint64_t rdata = (((uint64_t)rdata6 << 56) | ((uint64_t)rdata7 << 24) | ((uint64_t)rdata8 >> 8)) + nonce;

  rdata6 = (uint32_t)(rdata >> 56) | (rdata6 & 0xFFFFFF00);
  rdata7 = (uint32_t)(rdata >> 24);

  uint32_t rdata10 = (uint32_t)(rdata << 8) | 0x80;
  rdata8 = (uint32_t)(rdata << 8) | (rdata8 & 0xFF);

  data.v[0x0] = rdata6;
  data.v[0x1] = rdata7;
  data.v[0x2] = rdata8;
  data.v[0x3] = c_msg.pseed[0];
  data.v[0x4] = c_msg.pseed[1];
  data.v[0x5] = c_msg.pseed[2];
  data.v[0x6] = thrdata->rdata[3];
  data.v[0x7] = thrdata->rdata[4];
  data.v[0x8] = thrdata->rdata[5];
  data.v[0x9] = thrdata->rdata[6];
  data.v[0xA] = thrdata->rdata[7];
  data.v[0xB] = thrdata->rdata[8];
  data.v[0xC] = rdata6;
  data.v[0xD] = rdata7;
  data.v[0xE] = rdata10;
  data.v[0xF] = 0x00000000;

  /*if (vcpu == 0 && idx == 0)
		printf(
			"%04x %04x %04x %04x %04x %04x %04x %04x\n%04x %04x %04x %04x %04x %04x %04x %04x\n\n",
			data.v[0], data.v[1], data.v[2], data.v[3], data.v[4], data.v[5], data.v[6], data.v[7],
			data.v[8], data.v[9], data.v[10], data.v[11], data.v[12], data.v[13], data.v[14], data.v[15]
		);*/

  state = sha256_transform(data, state);

  data.v[0x0] = 0x00000000;
  data.v[0x1] = 0x00000000;
  data.v[0x2] = 0x00000000;
  data.v[0x3] = 0x00000000;
  data.v[0x4] = 0x00000000;
  data.v[0x5] = 0x00000000;
  data.v[0x6] = 0x00000000;
  data.v[0x7] = 0x00000000;
  data.v[0x8] = 0x00000000;
  data.v[0x9] = 0x00000000;
  data.v[0xA] = 0x00000000;
  data.v[0xB] = 0x00000000;
  data.v[0xC] = 0x00000000;
  data.v[0xD] = 0x00000000;
  data.v[0xE] = 0x00000000;
  data.v[0xF] = 0x000003d8;

  /*if (vcpu == 0 && idx == 0)
		printf(
			"%04x %04x %04x %04x %04x %04x %04x %04x\n%04x %04x %04x %04x %04x %04x %04x %04x\n\n",
			data.v[0], data.v[1], data.v[2], data.v[3], data.v[4], data.v[5], data.v[6], data.v[7],
			data.v[8], data.v[9], data.v[10], data.v[11], data.v[12], data.v[13], data.v[14], data.v[15]
		);*/

  state = sha256_transform(data, state);

  /*if (vcpu == 0 && idx == 0)
		printf(
			"%04x %04x %04x %04x %04x %04x %04x %04x\n\n",
			state.v[0], state.v[1], state.v[2], state.v[3], state.v[4], state.v[5], state.v[6], state.v[7]
		);*/

  for (int i = 0; i < 8; i += 1) {
    if (state.v[i] > c_msg.target.v[i])
      return;

    if (state.v[i] < c_msg.target.v[i]) {
      if (atomicAdd(&result->found, 1) == 0)
        result->nonce = nonce, result->vcpu = vcpu;

      return;
    }
  }
}

static DevHashResult *d_result;

__host__ void bitcredit_cpu_init(uint32_t gpu_id, uint32_t cpu_id, uint64_t threads) {
  (void)gpu_id, (void)cpu_id, (void)threads;
  CUDA_CALL_OR_RET(hipMalloc(&d_result, sizeof(*d_result)));
}

__host__ bool bitcredit_setBlockTarget(uint32_t gpu_id, uint32_t gpu_threads, uint32_t cpu_id, uint32_t expired,
                                       const unsigned char *data, const void *target, const unsigned char *rdata) {
  (void)gpu_id, (void)cpu_id;

  /*printf("gpu_threads: %u, expired: %04x\n", (unsigned)gpu_threads, (unsigned)expired);
	printf("data:\n");

	unsigned char padded_data[128];
	memset(padded_data, 0, sizeof(padded_data));
	memcpy(padded_data, data, 123);

	for (int i = 0; i < 32; i += 1)
	{
		for (int j = 0; j < 4; j += 1)
			printf("%02hhx ", padded_data[i * 4 + j]);
		printf("\n");
	}

	printf("target:\n");
	for (int i = 0; i < 32; i += 1)
		printf("%02hhx ", ((const unsigned char *)target)[i]);
	printf("\n");

	printf("rdata:\n");
	for (unsigned i = 0; i < gpu_threads; i += 1)
	{
		for (int j = 0; j < 32; j += 1)
			printf("%02hhx ", rdata[i * 32 + j]);
		printf("\n");
	}*/

  MsgData msg;

  for (int i = 0; i < 3; i += 1)
    be32enc(&msg.pseed[i], *((const uint32_t *)(data + 76) + i));

  for (int i = 0; i < 8; i += 1)
    be32enc(&msg.target.v[i], *((const uint32_t *)target + i));

  for (uint32_t gpu_thread = 0; gpu_thread < gpu_threads; gpu_thread += 1) {
    ThreadData *thrdata = &msg.thrdata[gpu_thread];
    const unsigned char *thr_rdata = rdata + gpu_thread * 32;

    vec8u state = h256;
    vec16u shadata;

    be32enc(&shadata.v[0x0], *((const uint32_t *)data + 0x0));

    uint32_t word;
    be32enc(&word, *((const uint32_t *)data + 0x1));

    shadata.v[0x1] = (word & 0xFFFFFF00) | (expired >> 24);
    shadata.v[0x2] = (expired << 8) | data[11];

    for (int i = 0x3; i <= 0x9; i += 1)
      be32enc(&shadata.v[i], *((const uint32_t *)data + i));

    be32enc(&word, *((const uint32_t *)data + 0xA));
    shadata.v[0xA] = (word & 0xFFFFFF00) | thr_rdata[0];

    for (int i = 0xB, j = 0; i <= 0xF; i += 1, j += 1)
      be32enc(&shadata.v[i], *((const uint32_t *)(thr_rdata + 1) + j));

    thrdata->state = sha256_transform(shadata, state);

    for (int i = 0x0, j = 0; i <= 0x1; i += 1, j += 1)
      be32enc(&thrdata->rdata[i], *((const uint32_t *)(thr_rdata + 21) + j));

    be32enc(&word, *(const uint32_t *)(thr_rdata + 28));
    thrdata->rdata[0x2] = (word << 8) | data[75];

    be32enc(&word, *((const uint32_t *)data + 0x16));
    thrdata->rdata[0x3] = (word & 0xFFFFFF00) | thr_rdata[0];

    for (int i = 0x4, j = 0; i <= 0x8; i += 1, j += 1)
      be32enc(&thrdata->rdata[i], *((const uint32_t *)(thr_rdata + 1) + j));
  }

  CUDA_CALL_OR_RET_X(hipMemcpyToSymbol(HIP_SYMBOL(c_msg), &msg, sizeof(msg)), false);
  return true;
}

__host__ HashResult bitcredit_cpu_hash(uint32_t gpu_id, uint32_t cpu_id, uint32_t gpu_threads, uint64_t threads,
                                       uint64_t start_nonce, uint32_t expired) {
  (void)gpu_id, (void)expired;

  HashResult r;
  r.nonce = UINT64_MAX, r.vcpu = UINT64_MAX, r.cpu_id = cpu_id;

  DevHashResult devresult;
  devresult.nonce = UINT64_MAX, devresult.vcpu = UINT32_MAX, devresult.found = 0;

  CUDA_CALL_OR_RET_X(hipMemcpy(d_result, &devresult, sizeof(devresult), hipMemcpyHostToDevice), r);

  unsigned threads_per_block = 512;
  dim3 block(threads_per_block);

  unsigned threads_per_gpu_threads_block = threads_per_block * gpu_threads;

  dim3 grid((unsigned)((threads + threads_per_gpu_threads_block - 1) / threads_per_gpu_threads_block), gpu_threads);

  bitcredit_gpu_hash<<<grid, block>>>(start_nonce / gpu_threads, d_result);

  CUDA_CALL_OR_RET_X(hipDeviceSynchronize(), r);
  CUDA_CALL_OR_RET_X(hipMemcpy(&devresult, d_result, sizeof(devresult), hipMemcpyDeviceToHost), r);

  r.nonce = devresult.nonce;
  r.vcpu = (devresult.vcpu == UINT32_MAX) ? UINT64_MAX : devresult.vcpu;

  return r;
}
